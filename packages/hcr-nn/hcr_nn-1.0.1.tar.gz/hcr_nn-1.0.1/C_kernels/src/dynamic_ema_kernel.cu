#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// EMA kernel: update each a[i,j,k]
__global__ void ema_update_kernel(
    const float* __restrict__ x,
    const float* __restrict__ y,
    const float* __restrict__ z,
    float* __restrict__ a,
    float ema_lambda,
    int D
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = D * D * D;

    if (idx >= total) return;

    int i = idx / (D * D);
    int j = (idx / D) % D;
    int k = idx % D;

    float update_val = x[i] * y[j] * z[k];
    float old_val = a[idx];
    a[idx] = (1.0f - ema_lambda) * old_val + ema_lambda * update_val;
}


torch::Tensor ema_update_cuda(
    torch::Tensor a,   // (D,D,D)
    torch::Tensor x,   // (D,)
    torch::Tensor y,   // (D,)
    torch::Tensor z,   // (D,)
    float ema_lambda
) {
    int D = x.size(0);
    int total = D * D * D;

    const int threads = 256;
    const int blocks = (total + threads - 1) / threads;

    ema_update_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        y.data_ptr<float>(),
        z.data_ptr<float>(),
        a.data_ptr<float>(),
        ema_lambda,
        D
    );

    return a;
}
