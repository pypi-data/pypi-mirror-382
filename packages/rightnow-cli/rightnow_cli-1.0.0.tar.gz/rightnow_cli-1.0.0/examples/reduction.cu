
#include <hip/hip_runtime.h>
// Parallel reduction kernel - many optimization opportunities
__global__ void reduce_sum(float *g_idata, float *g_odata, unsigned int n) {
    extern __shared__ float sdata[];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data to shared memory
    sdata[tid] = (i < n) ? g_idata[i] : 0;
    __syncthreads();
    
    // Reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Write result for this block
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}